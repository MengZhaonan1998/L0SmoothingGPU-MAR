#include "hip/hip_runtime.h"
#include "header.h"


__global__ void SetSingleValueKernel(hipfftReal* d_fpInputData, hipfftReal value, size_t nIdx)
{
	int tid = threadIdx.x;
	if (tid == 0)
	{
		d_fpInputData[nIdx] = value;
	}
}


__global__ void ThresholdKernel(hipfftReal* d_Array, hipfftReal threshold, size_t nSize)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < nSize)
	{
		if (d_Array[tid] < threshold)
			d_Array[tid] = threshold;
	}
}


__global__ void InitVectorKernel(hipfftReal* fVector, hipfftReal fValue, size_t nSize)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < nSize) {
		fVector[tid] = fValue;
	}
}


__global__ void Denormin2Kernel(hipfftReal* d_Denormin2, hipfftComplex* d_otfFx, hipfftComplex* d_otfFy, size_t nRows, size_t nCols)
{
	int tx = threadIdx.x + blockDim.x * blockIdx.x;
	int ty = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = tx * nCols + ty;
	hipfftReal absOtfFx, absOtfFy;
	
	if (tx < nRows && ty < nCols)
	{
		absOtfFx = d_otfFx[idx].x * d_otfFx[idx].x + d_otfFx[idx].y * d_otfFx[idx].y;
		absOtfFy = d_otfFy[idx].x * d_otfFy[idx].x + d_otfFy[idx].y * d_otfFy[idx].y;
		d_Denormin2[idx] = absOtfFx + absOtfFy;
	}
}


__global__ void HSubproblemKernel(hipfftReal* d_fpInputData, hipfftReal* d_hArray, size_t nRows, size_t nCols)
{
	int tx = threadIdx.x + blockDim.x * blockIdx.x;
	int ty = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = tx * nCols + ty;

	if (tx < nRows && ty < nCols)
	{
		if (ty == (nCols - 1)) {
			d_hArray[idx] = d_fpInputData[tx * nCols] - d_fpInputData[idx];
		}
		else {
			d_hArray[idx] = d_fpInputData[tx * nCols + ty + 1] - d_fpInputData[idx];
		}
	}
}


__global__ void VSubproblemKernel(hipfftReal* d_fpInputData, hipfftReal* d_vArray, size_t nRows, size_t nCols)
{
	int tx = threadIdx.x + blockDim.x * blockIdx.x;
	int ty = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = tx * nCols + ty;

	if (tx < nRows && ty < nCols)
	{
		if (tx == (nRows - 1)) {
			d_vArray[idx] = d_fpInputData[ty] - d_fpInputData[idx];
		}
		else {
			d_vArray[idx] = d_fpInputData[(tx + 1) * nCols + ty] - d_fpInputData[idx];
		}
	}
}


__global__ void HvValueConditionKernel(hipfftReal* d_h, hipfftReal* d_v, size_t nSize, hipfftReal fLambda, hipfftReal fBeta)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	hipfftReal temp;
	if (tid < nSize)
	{
		temp = d_h[tid] * d_h[tid] + d_v[tid] * d_v[tid];
		if (temp < fLambda / fBeta)
		{
			d_h[tid] = 0;
			d_v[tid] = 0;
		}
	}
}


__global__ void SSubproblemKernel_h(hipfftReal* d_Array, hipfftReal* d_h, size_t nRows, size_t nCols)
{
	int tx = threadIdx.x + blockDim.x * blockIdx.x;
	int ty = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = tx * nCols + ty;
	if (tx < nRows && ty < nCols)
	{
		if (ty == 0) {
			d_Array[idx] = d_h[tx * nCols + nCols - 1] - d_h[idx];
		}
		else {
			d_Array[idx] = d_h[tx * nCols + ty - 1] - d_h[idx];
		}
	}
}


__global__ void SSubproblemKernel_v(hipfftReal* d_Array, hipfftReal* d_v, size_t nRows, size_t nCols)
{
	int tx = threadIdx.x + blockDim.x * blockIdx.x;
	int ty = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = tx * nCols + ty;
	if (tx < nRows && ty < nCols)
	{
		if (tx == 0) {
			d_Array[idx] = d_v[(nRows - 1) * nCols + ty] - d_v[idx];
		}
		else {
			d_Array[idx] = d_v[(tx -1) * nCols + ty] - d_v[idx];
		}
	}
}


__global__ void SSubproblemKernel_FS(hipfftComplex* d_FS, hipfftComplex* d_Normin1, hipfftReal* d_Denormin, hipfftReal beta, size_t nSize)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;	
	hipfftComplex temp_FS, temp_Normin1;
	hipfftReal temp_Denormin;

	if (tid < nSize)
	{
		temp_FS = d_FS[tid];
		temp_Normin1 = d_Normin1[tid];
		temp_Denormin = d_Denormin[tid];

		temp_FS.x = (beta * temp_FS.x + temp_Normin1.x) / temp_Denormin;
		temp_FS.y = (beta * temp_FS.y + temp_Normin1.y) / temp_Denormin;
	
		d_FS[tid] = temp_FS;
	}
}


void L0Smoothing(float* h_fpInputData, size_t nRows, size_t nCols, float fLambda, float fKappa)
{
	auto t_start = std::chrono::high_resolution_clock::now();
	float betamax = 1e5;
	size_t nArraySize = nRows * nCols;
	size_t nFFTArraySize = nRows * (nCols / 2 + 1);
	hipfftReal threshold = 0.0;
	hipError_t cudaStatus;
	hipfftHandle cufftPlan1;
	hipfftHandle cufftPlan2;
	checkCufftErrors(hipfftPlan2d(&cufftPlan1, nRows, nCols, HIPFFT_R2C)); // FFT2D
	checkCufftErrors(hipfftPlan2d(&cufftPlan2, nRows, nCols, HIPFFT_C2R));

	// Copy input data from host to device
	hipfftReal* d_fpInputData;
	checkCudaErrors(hipMalloc((void**)&d_fpInputData, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMemcpy(d_fpInputData, h_fpInputData, sizeof(hipfftReal) * nArraySize, hipMemcpyHostToDevice));
	
	// Allocation/Initialization d_fx/d_fy
	hipfftReal* d_fx, * d_fy;
	checkCudaErrors(hipMalloc((void**)&d_fx, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMalloc((void**)&d_fy, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMemset(d_fx, 0, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMemset(d_fy, 0, sizeof(hipfftReal) * nArraySize));
	
	// Allocation d_otfFx/d_otfFy
	hipfftComplex* d_otfFx, * d_otfFy;
	checkCudaErrors(hipMalloc((void**)&d_otfFx, sizeof(hipfftComplex) * nFFTArraySize));
	checkCudaErrors(hipMalloc((void**)&d_otfFy, sizeof(hipfftComplex) * nFFTArraySize));

	// Allocation d_Normin1/d_Normin2/d_Normin2_temp/d_Denormin/d_Denormin2
	hipfftComplex* d_Normin1;
	hipfftReal* d_Denormin2, * d_Denormin, * d_Normin2, * d_Normin2_temp;
	checkCudaErrors(hipMalloc((void**)&d_Normin1, sizeof(hipfftComplex) * nFFTArraySize));
	checkCudaErrors(hipMalloc((void**)&d_Normin2, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMalloc((void**)&d_Normin2_temp, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMalloc((void**)&d_Denormin, sizeof(hipfftReal) * nFFTArraySize));
	checkCudaErrors(hipMalloc((void**)&d_Denormin2, sizeof(hipfftReal) * nFFTArraySize));

	// Allocation d_h/d_v
	hipfftReal* d_h, * d_v;
	checkCudaErrors(hipMalloc((void**)&d_h, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMalloc((void**)&d_v, sizeof(hipfftReal) * nArraySize));

	// Allocation d_FS
	hipfftComplex* d_FS;
	checkCudaErrors(hipMalloc((void**)&d_FS, sizeof(hipfftComplex) * nFFTArraySize));

	// Compute the max value of the input data
	int maxIndex = hipblasIsamax(nArraySize, d_fpInputData, 1);   // Fairly slow (?)	   //std::sort(h_fpInputData, h_fpInputData + nArraySize);  // 20 ms
	hipfftReal maxValue = h_fpInputData[maxIndex];
	hipblasSscal(nArraySize, 1.0f / maxValue, d_fpInputData, 1);
	//ScaleVectorKernel<<<(nArraySize + 63) / 64, 64 >>>(d_fpInputData, 1.0f / maxValue, nArraySize);
	
	// PseudoPsf2Otf 
	SetSingleValueKernel << <1, 1 >> > (d_fx, -1, 0); checkCudaErrors(hipGetLastError());
	SetSingleValueKernel << <1, 1 >> > (d_fx, 1, nCols-1); checkCudaErrors(hipGetLastError());
	SetSingleValueKernel << <1, 1 >> > (d_fy, -1, 0); checkCudaErrors(hipGetLastError());
	SetSingleValueKernel << <1, 1 >> > (d_fy, 1, nCols * (nRows - 1)); checkCudaErrors(hipGetLastError());
	
	checkCufftErrors(hipfftExecR2C(cufftPlan1, d_fx, d_otfFx));
	checkCufftErrors(hipfftExecR2C(cufftPlan1, d_fy, d_otfFy));
	
	// Compute Normin1: FFT input data
	checkCufftErrors(hipfftExecR2C(cufftPlan1, d_fpInputData, d_Normin1));  
	
	// Compute Denomin2
	dim3 threads(16, 16, 1);
	dim3 blocks((nRows + threads.x - 1) / threads.x, (nCols + threads.y - 1) / threads.y, 1);	
	Denormin2Kernel << <blocks, threads >> > (d_Denormin2, d_otfFx, d_otfFy, nRows, nCols / 2 + 1); checkCudaErrors(hipGetLastError());

	float beta = 2 * fLambda;
	while (beta < betamax)
	{
		// Denorm = 1 + beta * Denormin2
		InitVectorKernel<< < (nFFTArraySize + 63) / 64, 64 >> > (d_Denormin, 1.0, nFFTArraySize); checkCudaErrors(hipGetLastError());
		hipblasSaxpy(nFFTArraySize, beta, d_Denormin2, 1, d_Denormin, 1);

		// h-v subproblem
		HSubproblemKernel << <blocks, threads >> > (d_fpInputData, d_h, nRows, nCols); checkCudaErrors(hipGetLastError());
		VSubproblemKernel << <blocks, threads >> > (d_fpInputData, d_v, nRows, nCols); checkCudaErrors(hipGetLastError());

		// h/v condition
		HvValueConditionKernel << <(nArraySize + 63) / 64, 64 >> > (d_h, d_v, nArraySize, fLambda, beta); checkCudaErrors(hipGetLastError());

		// S subproblem
		SSubproblemKernel_h << < blocks, threads >> > (d_Normin2, d_h, nRows, nCols); checkCudaErrors(hipGetLastError());	
		SSubproblemKernel_v << < blocks, threads >> > (d_Normin2_temp, d_v, nRows, nCols); checkCudaErrors(hipGetLastError());	
		hipblasSaxpy(nArraySize, 1.0, d_Normin2_temp, 1, d_Normin2, 1);
		checkCufftErrors(hipfftExecR2C(cufftPlan1, d_Normin2, d_FS));	
		SSubproblemKernel_FS << <(nFFTArraySize + 63) / 64, 64 >> > (d_FS, d_Normin1, d_Denormin, beta, nFFTArraySize);

		// IFFT
		checkCufftErrors(hipfftExecC2R(cufftPlan2, d_FS, d_fpInputData));
		hipblasSscal(nArraySize, 1.0f/nArraySize, d_fpInputData, 1);
		
		// threshold
		//ThresholdKernel << <(nArraySize + 63) / 64, 64 >> > (d_fpInputData, threshold, nArraySize);

		beta *= fKappa;
		std::cout << ".";
	}
	
	hipblasSscal(nArraySize, maxValue, d_fpInputData, 1);

	checkCudaErrors(hipMemcpy(h_fpInputData, d_fpInputData, sizeof(hipfftReal) * nArraySize, hipMemcpyDeviceToHost));

	//hipfftReal* h_output = new hipfftReal[nFFTArraySize];
	//hipMemcpy(h_output, d_Denormin2, sizeof(hipfftReal) * nFFTArraySize, hipMemcpyDeviceToHost);
	//outputRealMatrix<hipfftReal>(h_output, 1, 5);
	//hipfftComplex* h_output = new hipfftComplex[nFFTArraySize];
	//hipMemcpy(h_output, d_Normin1, sizeof(hipfftComplex) * nFFTArraySize, hipMemcpyDeviceToHost);
	//outputComplexMatrix(h_output, 1, 5);	

	// Resource relief
	hipfftDestroy(cufftPlan1);
	hipfftDestroy(cufftPlan2);
	checkCudaErrors(hipFree(d_fpInputData));
	checkCudaErrors(hipFree(d_fx));
	checkCudaErrors(hipFree(d_fy));
	hipFree(d_otfFx);
	hipFree(d_otfFy);
	hipFree(d_Normin1);
	hipFree(d_Normin2);
	hipFree(d_Normin2_temp);
	hipFree(d_Denormin);
	hipFree(d_Denormin2);
	hipFree(d_h);
	hipFree(d_v);
	hipFree(d_FS);
	
	auto t_end = std::chrono::high_resolution_clock::now();
	auto ms_duration = std::chrono::duration_cast<std::chrono::milliseconds>(t_end - t_start);
	std::cout << "L0Smoothing took " << ms_duration.count() << " ms" << std::endl;
	return;
}


void L0SmoothingMultiSlice(float* h_fpInputData, size_t nRows, size_t nCols, size_t nSlices, float fLambda, float fKappa)
{
	auto t_start = std::chrono::high_resolution_clock::now();
	float betamax = 1e5;
	size_t nArraySize = nRows * nCols;
	size_t nFFTArraySize = nRows * (nCols / 2 + 1);
	hipError_t cudaStatus;
	hipfftHandle cufftPlan1;
	hipfftHandle cufftPlan2;
	checkCufftErrors(hipfftPlan2d(&cufftPlan1, nRows, nCols, HIPFFT_R2C)); // FFT2D
	checkCufftErrors(hipfftPlan2d(&cufftPlan2, nRows, nCols, HIPFFT_C2R)); 

	// Allocation d_fpInputData
	hipfftReal* d_fpInputData;
	checkCudaErrors(hipMalloc((void**)&d_fpInputData, sizeof(hipfftReal) * nArraySize));

	// Allocation/Initialization d_fx/d_fy
	hipfftReal* d_fx, * d_fy;
	checkCudaErrors(hipMalloc((void**)&d_fx, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMalloc((void**)&d_fy, sizeof(hipfftReal) * nArraySize));

	// Allocation d_otfFx/d_otfFy
	hipfftComplex* d_otfFx, * d_otfFy;
	checkCudaErrors(hipMalloc((void**)&d_otfFx, sizeof(hipfftComplex) * nFFTArraySize));
	checkCudaErrors(hipMalloc((void**)&d_otfFy, sizeof(hipfftComplex) * nFFTArraySize));

	// Allocation d_Normin1/d_Normin2/d_Normin2_temp/d_Denormin/d_Denormin2
	hipfftComplex* d_Normin1;
	hipfftReal* d_Denormin2, * d_Denormin, * d_Normin2, * d_Normin2_temp;
	checkCudaErrors(hipMalloc((void**)&d_Normin1, sizeof(hipfftComplex) * nFFTArraySize));
	checkCudaErrors(hipMalloc((void**)&d_Normin2, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMalloc((void**)&d_Normin2_temp, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMalloc((void**)&d_Denormin, sizeof(hipfftReal) * nFFTArraySize));
	checkCudaErrors(hipMalloc((void**)&d_Denormin2, sizeof(hipfftReal) * nFFTArraySize));

	// Allocation d_h/d_v
	hipfftReal* d_h, * d_v;
	checkCudaErrors(hipMalloc((void**)&d_h, sizeof(hipfftReal) * nArraySize));
	checkCudaErrors(hipMalloc((void**)&d_v, sizeof(hipfftReal) * nArraySize));

	// Allocation d_FS
	hipfftComplex* d_FS;
	checkCudaErrors(hipMalloc((void**)&d_FS, sizeof(hipfftComplex) * nFFTArraySize));

	for (int i = 0; i < nSlices; i++)
	{
		std::cout << "Slice " << i << ": ";
		auto t_0 = std::chrono::high_resolution_clock::now();
		float betamax = 1e5;

		checkCudaErrors(hipMemcpy(d_fpInputData, h_fpInputData + i * nCols * nRows, sizeof(hipfftReal) * nArraySize, hipMemcpyHostToDevice));

		// Compute the max value of the input data
		int maxIndex = hipblasIsamax(nArraySize, d_fpInputData, 1);   // Fairly slow (?)	   //std::sort(h_fpInputData, h_fpInputData + nArraySize);  // 20 ms
		hipfftReal maxValue = h_fpInputData[i * nCols * nRows + maxIndex];
		hipblasSscal(nArraySize, 1.0f / maxValue, d_fpInputData, 1);
		//ScaleVectorKernel<<<(nArraySize + 63) / 64, 64 >>>(d_fpInputData, 1.0f / maxValue, nArraySize);

		// PseudoPsf2Otf 
		checkCudaErrors(hipMemset(d_fx, 0, sizeof(hipfftReal) * nArraySize));
		checkCudaErrors(hipMemset(d_fy, 0, sizeof(hipfftReal) * nArraySize));
		SetSingleValueKernel << <1, 1 >> > (d_fx, -1, 0); checkCudaErrors(hipGetLastError());
		SetSingleValueKernel << <1, 1 >> > (d_fx, 1, nCols - 1); checkCudaErrors(hipGetLastError());
		SetSingleValueKernel << <1, 1 >> > (d_fy, -1, 0); checkCudaErrors(hipGetLastError());
		SetSingleValueKernel << <1, 1 >> > (d_fy, 1, nCols * (nRows - 1)); checkCudaErrors(hipGetLastError());
		
		checkCufftErrors(hipfftExecR2C(cufftPlan1, d_fx, d_otfFx));
		checkCufftErrors(hipfftExecR2C(cufftPlan1, d_fy, d_otfFy));

		// Compute Normin1: FFT input data
		checkCufftErrors(hipfftExecR2C(cufftPlan1, d_fpInputData, d_Normin1));

		// Compute Denomin2
		dim3 threads(16, 16, 1);
		dim3 blocks((nRows + threads.x - 1) / threads.x, (nCols + threads.y - 1) / threads.y, 1);
		Denormin2Kernel << <blocks, threads >> > (d_Denormin2, d_otfFx, d_otfFy, nRows, nCols / 2 + 1); checkCudaErrors(hipGetLastError());

		float beta = 2 * fLambda;
		while (beta < betamax)
		{
			// Denorm = 1 + beta * Denormin2
			InitVectorKernel<< < (nFFTArraySize + 63) / 64, 64 >> > (d_Denormin, 1.0, nFFTArraySize); checkCudaErrors(hipGetLastError());
			hipblasSaxpy(nFFTArraySize, beta, d_Denormin2, 1, d_Denormin, 1);

			// h-v subproblem
			HSubproblemKernel << <blocks, threads >> > (d_fpInputData, d_h, nRows, nCols); checkCudaErrors(hipGetLastError());
			VSubproblemKernel << <blocks, threads >> > (d_fpInputData, d_v, nRows, nCols); checkCudaErrors(hipGetLastError());

			// h/v condition
			HvValueConditionKernel << <(nArraySize + 63) / 64, 64 >> > (d_h, d_v, nArraySize, fLambda, beta); checkCudaErrors(hipGetLastError());

			// S subproblem
			SSubproblemKernel_h << < blocks, threads >> > (d_Normin2, d_h, nRows, nCols); checkCudaErrors(hipGetLastError());
			SSubproblemKernel_v << < blocks, threads >> > (d_Normin2_temp, d_v, nRows, nCols); checkCudaErrors(hipGetLastError());
			hipblasSaxpy(nArraySize, 1.0, d_Normin2_temp, 1, d_Normin2, 1);
			checkCufftErrors(hipfftExecR2C(cufftPlan1, d_Normin2, d_FS));
			SSubproblemKernel_FS << <(nFFTArraySize + 63) / 64, 64 >> > (d_FS, d_Normin1, d_Denormin, beta, nFFTArraySize); checkCudaErrors(hipGetLastError());

			// IFFT
			checkCufftErrors(hipfftExecC2R(cufftPlan2, d_FS, d_fpInputData));
			hipblasSscal(nArraySize, 1.0f / nArraySize, d_fpInputData, 1);

			beta *= fKappa;
			std::cout << ".";	
		}

		hipblasSscal(nArraySize, maxValue, d_fpInputData, 1);

		checkCudaErrors(hipMemcpy(h_fpInputData + i * nCols * nRows, d_fpInputData, sizeof(hipfftReal) * nArraySize, hipMemcpyDeviceToHost));

		auto t_1 = std::chrono::high_resolution_clock::now();
		auto ms_duration = std::chrono::duration_cast<std::chrono::milliseconds>(t_1 - t_0);
		std::cout << ms_duration.count() << " ms" << std::endl;
	}

	// Resource relief
	hipfftDestroy(cufftPlan1);
	hipfftDestroy(cufftPlan2);
	checkCudaErrors(hipFree(d_fpInputData));
	checkCudaErrors(hipFree(d_fx));
	checkCudaErrors(hipFree(d_fy));
	hipFree(d_otfFx);
	hipFree(d_otfFy);
	hipFree(d_Normin1);
	hipFree(d_Normin2);
	hipFree(d_Normin2_temp);
	hipFree(d_Denormin);
	hipFree(d_Denormin2);
	hipFree(d_h);
	hipFree(d_v);
	hipFree(d_FS);

	auto t_end = std::chrono::high_resolution_clock::now();
	auto ms_duration = std::chrono::duration_cast<std::chrono::milliseconds>(t_end - t_start);
	std::cout << "L0SmoothingMultiSlice took " << ms_duration.count() << " ms" << std::endl;
	return;
}
